#include "hip/hip_runtime.h"
#include<stdio.h>
#include"scrImagePgmPpmPackage.h"
#include "gputimer.h"

//Kernel which calculate the resized image
__global__ void createResizedImage(unsigned char *imageScaledData, int scaled_width, float scale_factor, hipTextureObject_t texObj)
{
	const unsigned int tidX = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int tidY = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned index = tidY*scaled_width+tidX;
       	
	// Step 4: Read the texture memory from your texture reference in CUDA Kernel
	imageScaledData[index] = tex2D<unsigned char>(texObj,(float)(tidX*scale_factor),(float)(tidY*scale_factor));
}

int main(int argc, char*argv[])
{
	// Scaling ratio, Block size, Grid size
	int height=0, width =0, scaled_height=0,scaled_width=0;
	//Define the scaling ratio	
	float scaling_ratio=0.5;

	if (argc <= 1) 
		printf("Please enter img path!\n");

	char *img_name = argv[1];

	if(argc > 2)
		sscanf(argv[2],"%f",&scaling_ratio);

	int block_size = 32;

	if (argc > 3) 
        block_size = atoi(argv[3]);

	GpuTimer timer;
    timer.Start();

	unsigned char*data;
	unsigned char*scaled_data,*d_scaled_data;

	char *inputStr = img_name;
	char outputStr[1024] = {"scaled.pgm"};
	hipError_t returnValue;

	//Create a channel Description to be used while linking to the tecture
	hipArray* cu_array;
	hipChannelFormatKind kind = hipChannelFormatKindUnsigned;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, kind);

	get_PgmPpmParams(inputStr, &height, &width);	//getting height and width of the current image
	data = (unsigned char*)malloc(height*width*sizeof(unsigned char));
	// printf("\n Reading image width height and width [%d][%d]", height, width);
	scr_read_pgm( inputStr , data, height, width );//loading an image to "inputimage"

	scaled_height = (int)(height*scaling_ratio);
	scaled_width = (int)(width*scaling_ratio);
	scaled_data = (unsigned char*)malloc(scaled_height*scaled_width*sizeof(unsigned char));
	// printf("\n scaled image width height and width [%d][%d]", scaled_height, scaled_width);

	//Allocate CUDA Array
 	returnValue = hipMallocArray( &cu_array, &channelDesc, width, height);
	// returnValue = (hipError_t)(returnValue | hipMemcpy( cu_array, data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice));
	returnValue = (hipError_t)(returnValue | hipMemcpyToArray( cu_array, 0, 0, data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice));

	if(returnValue != hipSuccess)
		printf("\n Got error while running CUDA API Array Copy");

	// Step 1. Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cu_array;
	// Step 2. Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// Step 3: Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	if(returnValue != hipSuccess) 
		printf("\n Got error while running CUDA API Bind Texture");
	
	hipMalloc(&d_scaled_data, scaled_height*scaled_width*sizeof(unsigned char) );


	dim3 dimBlock(block_size, block_size,1);
	dim3 dimGrid(scaled_width/dimBlock.x + 1,scaled_height/dimBlock.y + 1,1);

	// printf("\n Launching grid with blocks [%d][%d] ", dimGrid.x,dimGrid.y);

	createResizedImage<<<dimGrid, dimBlock>>>(d_scaled_data,scaled_width,1/scaling_ratio, texObj);

	returnValue = (hipError_t)(returnValue | hipDeviceSynchronize());

	returnValue = (hipError_t)(returnValue |hipMemcpy (scaled_data , d_scaled_data, scaled_height*scaled_width*sizeof(unsigned char), hipMemcpyDeviceToHost ));
	if(returnValue != hipSuccess) 
		printf("\n Got error while running CUDA API kernel");

	// Step 5: Destroy texture object
	hipDestroyTextureObject(texObj);
	
	scr_write_pgm( outputStr, scaled_data, scaled_height, scaled_width, "####" ); //storing the image with the detections
		
	if(data != NULL)
		free(data);
	if(cu_array !=NULL)
		hipFreeArray(cu_array);
	if(scaled_data != NULL)
		free(scaled_data);
	if(d_scaled_data!=NULL)
		hipFree(d_scaled_data);

    timer.Stop();
	float timer_result = timer.Elapsed();

    // printf("image;scale;block_size;time\n");
    printf("%s;%f;%d;%f\n", img_name, scaling_ratio, block_size, timer_result);

	return 0;
}
