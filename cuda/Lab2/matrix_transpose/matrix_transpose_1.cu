#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include "gputimer.h"

// #define N 2048
#define BLOCK_SIZE 32 

__global__ void matrix_transpose_naive(int *input, int *output, int N) {

	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	int index = indexY * N + indexX;
	int transposedIndex = indexX * N + indexY;

    // this has discoalesced global memory store  
	output[transposedIndex] = input[index];

	// this has discoalesced global memore load
	// output[index] = input[transposedIndex];
}

__global__ void matrix_transpose_shared(int *input, int *output, int N) {

	__shared__ int sharedMemory [BLOCK_SIZE] [BLOCK_SIZE];

	// global index	
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;

	// transposed global memory index
	int tindexX = threadIdx.x + blockIdx.y * blockDim.x;
	int tindexY = threadIdx.y + blockIdx.x * blockDim.y;

	// local index
	int localIndexX = threadIdx.x;
	int localIndexY = threadIdx.y;

	int index = indexY * N + indexX;
	int transposedIndex = tindexY * N + tindexX;

	// reading from global memory in coalesed manner and performing tanspose in shared memory
	sharedMemory[localIndexX][localIndexY] = input[index];

	__syncthreads();

	// writing into global memory in coalesed fashion via transposed data in shared memory
	output[transposedIndex] = sharedMemory[localIndexY][localIndexX];
}

//basically just fills the array with index.
void fill_array(int *data, int N) {
	for(int idx=0;idx<(N*N);idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int N) {
	printf("\n Original Matrix::\n");
	for(int idx=0;idx<(N*N);idx++) {
		if(idx%N == 0)
			printf("\n");
		printf(" %d ",  a[idx]);
	}
	printf("\n Transposed Matrix::\n");
	for(int idx=0;idx<(N*N);idx++) {
		if(idx%N == 0)
			printf("\n");
		printf(" %d ",  b[idx]);
	}
}

int main(int argc, char *argv[]) {
	// argv = N ; Type ; Grid size ; Block size
	int N =0;

    if (argc > 1) {
        N = atoi(argv[1]);
    } else {
        printf("Please pass number of elements as command line argument.\n");
        exit(EXIT_FAILURE);
    }

    bool shared = false;
    if (argc > 2 && strcmp(argv[2], "shared") == 0){
		shared = true;
    }

	int the_block_size = BLOCK_SIZE;
	int the_grid_size = N / BLOCK_SIZE;

	if (argc > 3) 
        the_grid_size = atoi(argv[4]);

	if (argc > 4) 
        the_block_size = atoi(argv[3]);

	int *a, *b;
    int *d_a, *d_b; // device copies of a, b, c

	int size = N * N *sizeof(int);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a, N);
	b = (int *)malloc(size);

	GpuTimer timer;
    timer.Start();
	
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	dim3 blockSize(the_block_size,the_block_size,1);
	dim3 gridSize(the_grid_size,the_grid_size,1);

	if(!shared)
	matrix_transpose_naive<<<gridSize,blockSize>>>(d_a,d_b, N);
	
	if(shared)
	matrix_transpose_shared<<<gridSize,blockSize>>>(d_a,d_b, N);

	// Copy result back to host
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	// print_output(a,b);

    hipFree(d_a);
	hipFree(d_b); 

    timer.Stop();
	float timer_result = timer.Elapsed();

	// terminate memories
	free(a);
	free(b);

    // printf("size;time;type;grid_size;block_size\n");
    printf("%d, %f, %s, %d, %d\n", N, timer_result, (shared ? "shared" : "naive"), the_grid_size, the_block_size);

	return 0;
}
